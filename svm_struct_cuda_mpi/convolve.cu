#include "hip/hip_runtime.h"
/*
  convolution between hog features and hog templates
  author: Yu Xiang
  Date: 04/14/2011
*/

extern "C"
{
#include "convolve.h"
#include "matrix.h"
}
#include "cutil_inline.h"

#define BLOCK_SIZE 21

__constant__ float hog_template[2048];
__global__ void convolve2D(CUMATRIX C, CUMATRIX A, CUMATRIX B, int index);

CUMATRIX fconv(CUMATRIX A, CUMATRIX B)
{
  CUMATRIX A_device;
  CUMATRIX B_device;
  CUMATRIX C, C_device;

  A_device = alloc_device_cumatrix(A);
  B_device = alloc_device_cumatrix(B);

  // allocate hog response cumatrix
  C.dims_num = 2;
  C.dims = (int*)malloc(sizeof(int)*2);
  C.dims[0] = A.dims[0];
  C.dims[1] = A.dims[1];
  C.length = C.dims[0]*C.dims[1];
  C.data = (float*)malloc(sizeof(float)*C.length);
  C_device = alloc_device_cumatrix(C);
  cutilSafeCall(hipMemset(C_device.data, 0, sizeof(float)*C_device.length));

  /* setup execution parameters */
  dim3 threads(BLOCK_SIZE, BLOCK_SIZE+2*(B.dims[0]/2));
  dim3 grid((C.dims[1]+BLOCK_SIZE-1) / BLOCK_SIZE, (C.dims[0]+BLOCK_SIZE-1) / BLOCK_SIZE);

  for(int i = 0; i < B.dims[2]; i++)
  {
    // copy to constant memory
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(hog_template), B.data+i*B.dims[0]*B.dims[1], sizeof(float)*B.dims[0]*B.dims[1]));
    convolve2D<<< grid, threads >>>(C_device, A_device, B_device, i);
    hipDeviceSynchronize();
  }

  /* copy result from device to host */
  cutilSafeCall(hipMemcpy(C.data, C_device.data, sizeof(float)*C.length, hipMemcpyDeviceToHost) );

  free_device_cumatrix(&A_device);
  free_device_cumatrix(&B_device);
  free_device_cumatrix(&C_device);
  return C;
}

// implementation of the convolution algorithm described in nvidia
// Image convolution with CUDA for nonseperable kernel
__global__ void convolve2D(CUMATRIX C, CUMATRIX A, CUMATRIX B, int index)
{
  __shared__ float data[3*BLOCK_SIZE][3*BLOCK_SIZE];

  // template size
  int nx = B.dims[1];
  int ny = B.dims[0];

  // feature size
  int fx = A.dims[1];
  int fy = A.dims[0];

  // location in A.data of the current thread
  int x = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  int y = blockIdx.y*BLOCK_SIZE + threadIdx.y - ny/2;

  // load data
  float val;
  if(index == B.dims[2]-1)
    val = 1;
  else
    val = 0;

  int dx = x - BLOCK_SIZE;
  int dy = y;
  if(dx >= 0 && dx < fx && dy >= 0 && dy < fy)
    data[threadIdx.x][threadIdx.y] = A.data[index*fx*fy+dx*fy+dy];
  else
    data[threadIdx.x][threadIdx.y] = val;

  dx = x;
  dy = y;
  if(dx >= 0 && dx < fx && dy >= 0 && dy < fy)
    data[threadIdx.x+BLOCK_SIZE][threadIdx.y] = A.data[index*fx*fy+dx*fy+dy];
  else
    data[threadIdx.x+BLOCK_SIZE][threadIdx.y] = val;

  dx = x + BLOCK_SIZE;
  dy = y;
  if(dx >= 0 && dx < fx && dy >= 0 && dy < fy)
    data[threadIdx.x+2*BLOCK_SIZE][threadIdx.y] = A.data[index*fx*fy+dx*fy+dy];
  else
    data[threadIdx.x+2*BLOCK_SIZE][threadIdx.y] = val;
  __syncthreads();

  if(x < fx && y < fy && threadIdx.y >= ny/2 && threadIdx.y < ny/2 + BLOCK_SIZE)
  {
    // location in shared memory
    int xx = threadIdx.x + BLOCK_SIZE - nx/2;
    int yy = threadIdx.y - ny/2;
    float sum = 0;
    for(int i = 0; i < nx; i++)
    {
      for(int j = 0; j < ny; j++)
        sum += hog_template[i*ny+j] * data[xx+i][yy+j];
    }
    C.data[x*fy+y] += sum;
  }
}

/*
int main(int argc, char** argv)
{
  FILE *fp;
  MATRIX A, A_device;
  MATRIX B, B_device;
  MATRIX C, C_device;

  // load hog features
  fp = fopen(argv[1], "r");
  if(fp == NULL)
  {
    printf("can not open file %s\n", argv[1]);
    return 1;
  }
  A = read_matrix(fp);
  fclose(fp);
  A_device = alloc_device_matrix(A);

  // generate a random hog template
  B.dims_num = 3;
  B.dims = (int*)malloc(sizeof(int)*3);
  B.dims[0] = 16;
  B.dims[1] = 17;
  B.dims[2] = 32;
  B.length = 16*17*32;
  B.data = (float*)malloc(sizeof(float)*B.length);
  for(int i = 0; i < B.length; i++)
    B.data[i] = 1;
  B_device = alloc_device_matrix(B);

  // allocate hog response matrix
  C.dims_num = 2;
  C.dims = (int*)malloc(sizeof(int)*2);
  C.dims[0] = A.dims[0];
  C.dims[1] = A.dims[1];
  C.length = C.dims[0]*C.dims[1];
  C.data = (float*)malloc(sizeof(float)*C.length);
  C_device = alloc_device_matrix(C);
  cutilSafeCall(hipMemset(C_device.data, 0, sizeof(float)*C_device.length));

  // setup execution parameters
  dim3 threads(BLOCK_SIZE, BLOCK_SIZE+2*(B.dims[0]/2));
  dim3 grid((C.dims[1]+BLOCK_SIZE-1) / BLOCK_SIZE, (C.dims[0]+BLOCK_SIZE-1) / BLOCK_SIZE);

  unsigned int timer = 0;
  cutilCheckError(cutCreateTimer(&timer));
  cutilCheckError(cutStartTimer(timer));

  for(int i = 0; i < B.dims[2]; i++)
  {
    // copy to constant memory
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(hog_template), B.data+i*B.dims[0]*B.dims[1], sizeof(float)*B.dims[0]*B.dims[1]));
    convolve2D<<< grid, threads >>>(C_device, A_device, B_device, i);
    hipDeviceSynchronize();
  }

  // stop and destroy timer
  cutilCheckError(cutStopTimer(timer));
  float dSeconds = cutGetTimerValue(timer)/1000.0;
  cutilCheckError(cutDeleteTimer(timer));
  printf("time = %f\n", dSeconds);

  // copy result from device to host
  cutilSafeCall(hipMemcpy(C.data, C_device.data, sizeof(float)*C.length, hipMemcpyDeviceToHost) );

  fp = fopen(argv[2], "w");
  write_matrix(&C, fp);
  fclose(fp);

  free_device_matrix(&A_device);
  free_device_matrix(&B_device);
  free_device_matrix(&C_device);
  free_matrix(&A);
  free_matrix(&B);
  free_matrix(&C);
  return 0;
}
*/
