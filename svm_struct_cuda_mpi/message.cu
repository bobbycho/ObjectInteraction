#include "hip/hip_runtime.h"
/*
  compute messages in BP algorithm
  author: Yu Xiang
  Date: 04/21/2011
*/

extern "C"
{
#include "message.h"
#include "matrix.h"
}
#include "cutil_inline.h"

#define BLOCK_SIZE 8
#define MINUS_INFINITY -1.0E15

__constant__ float potential[8192];
__global__ void message(CUMATRIX M, int sbin, float dc, float ac, float wx, float wy);

void compute_message(CUMATRIX M, CUMATRIX V, int sbin, float dc, float ac, float wx, float wy)
{
  CUMATRIX M_device;

  /* allocate device memory */
  M_device = alloc_device_cumatrix(M);

  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(potential), V.data, sizeof(float)*V.length));

  // setup execution parameters
  dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid((M.dims[1]+BLOCK_SIZE-1) / BLOCK_SIZE, (M.dims[0]+BLOCK_SIZE-1) / BLOCK_SIZE);

  message<<< grid, threads >>>(M_device, sbin, dc, ac, wx, wy);
  hipDeviceSynchronize();

  // copy result from device to host
  cutilSafeCall(hipMemcpy(M.data, M_device.data, sizeof(float)*M.length, hipMemcpyDeviceToHost) );
  
  free_device_cumatrix(&M_device);
}

__global__ void message(CUMATRIX M, int sbin, float dc, float ac, float wx, float wy)
{
  int x, y, xi, yi, nx, ny;
  float max_val, val;

  nx = M.dims[1];
  ny = M.dims[0];
  x = blockIdx.x * blockDim.x + threadIdx.x;
  y = blockIdx.y * blockDim.y + threadIdx.y;

  if(x < nx && y < ny)
  {
    max_val = MINUS_INFINITY;
    for(xi = 0; xi < nx; xi++)
    {
      for(yi = 0; yi < ny; yi++)
      {
        val = potential[xi*ny+yi];
        val += powf(sbin*(xi-x) + dc*cosf(ac), 2.0) * wx;
        val += powf(sbin*(yi-y) + dc*sinf(ac), 2.0) * wy;
        if(val > max_val)
          max_val = val;
      }
    }
    M.data[x*ny+y] = max_val;
  }
}
